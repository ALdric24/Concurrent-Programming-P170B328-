#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <fstream>
#include "iostream"

//using namespace std;

typedef struct {
    char* name;
    int year;
    float grade;
} Student;

typedef struct {
    Student students[200];
    int numStudents;
} data_t;

//typedef struct {
//    char students[200];
//   
//} result;

char* file_buffer;
size_t file_size;
//__global__ void addResult(data_t data, data_t result);


void parseData(data_t& data, const char* file_buffer, size_t file_size) {

    data.numStudents = 200;
    char temp_name[256];
    int temp_year = 0;
    float temp_grade = 0.0;


    int i = 0;
    while (sscanf(file_buffer, "%[^,],%d,%f", temp_name, &temp_year, &temp_grade) == 3) {
        // Store the parsed data in the students array
        data.students[i].name = (char*)malloc(strlen(temp_name) + 1);
        strcpy(data.students[i].name, temp_name);
        data.students[i].year = temp_year;
        data.students[i].grade = temp_grade;

        printf("Name: %s, Score: %d, Score2: %f\n", data.students[i].name, data.students[i].year, data.students[i].grade);


        file_buffer = strchr(file_buffer, '\n') + 1;
        i++;
    }

    if (i < data.numStudents) {
        fprintf(stderr, "Error: failed to parse data from file buffer\n");
        return;
    }
}

data_t readfile(const char* textfile) {
    
    char* file_buffer;
    size_t file_size;
    FILE* file = fopen("data.txt", "r");
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file);
    file_buffer = static_cast<char*>(malloc(file_size));
    fread(file_buffer, 1, file_size, file);
    fclose(file);

    data_t data;
    data.numStudents = 200;
    parseData(data, file_buffer, file_size);

   
    free(file_buffer);

    return data;
}



void writeResultToFile(data_t result, const char* filename) {
	std::ofstream out_file(filename);
    if (out_file.is_open()) {
        for (int i = 0; i < result.numStudents; i++) {
            Student student = result.students[i];
            out_file << student.name << ", " << student.year << ", " << student.grade << std::endl;
        }
        out_file.close();
    }
}
__device__ bool checkNameFirstLetter(char letter) {

    return letter == 'C';
}
__global__ void addResult(data_t result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < result.numStudents) {
        Student student = result.students[idx];
        if (checkNameFirstLetter(student.name[0])) {
            // Convert name to uppercase
            int name_len = strlen(student.name);
            char* name = (char*)malloc(name_len + 1);
            for (int i = 0; i < name_len; i++) {
                name[i] = student.name[i];
                if (name[i] >= 'a' && name[i] <= 'z') {
                    name[i] = name[i] - 'a' + 'A';
                }
            }
            result.students[idx].name = name;
        }
        else {
            // Remove element from result data
            for (int i = idx; i < result.numStudents - 1; i++) {
                result.students[i] = result.students[i + 1];
            }
            result.numStudents = atomicAdd(&result.numStudents, -1);
        }
    }
}

//__global__ void addResult(data_t* data, data_t* result) {
//
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//
//    if (idx < data->numStudents) {
//
//        Student student = data->students[idx];
//
//
//        if (checkNameFirstLetter(student.name[0])) {
//
//            int numStudents = atomicAdd(&result->numStudents, 1);
//
//            result->students[numStudents].name = student.name;
//            result->students[numStudents].year = student.year;
//            result->students[numStudents].grade = student.grade;
//        }
//    }
//}
int main() {
    // Read data from file and copy to device memory
    data_t h_data = readfile("data.txt");
    data_t* d_data;
    hipMalloc(&d_data, sizeof(data_t));
    hipMemcpy(d_data, &h_data, sizeof(data_t), hipMemcpyHostToDevice);

    // Allocate and initialize result data on device memory
    data_t* d_result;
    hipMalloc(&d_result, sizeof(data_t));
    data_t h_result = { 0 };
    hipMemcpy(d_result, &h_data, sizeof(data_t), hipMemcpyHostToDevice);

    // Launch kernel to process data and store result
    addResult <<<2, 32 >>> (*d_result);

    // Copy result data from device memory to host memory
    hipMemcpy(&h_result, d_result, sizeof(data_t), hipMemcpyDeviceToHost);

    // Print result data
    for (int i = 0; i < h_result.numStudents; i++) {
        Student student = h_result.students[i];
        std::cout << student.name << ", " << student.year << ", " << student.grade << std::endl;
    }

    // Write result data to file
    writeResultToFile(h_result, "result.txt");

    // Free device memory
    hipFree(d_data);
    hipFree(d_result);

    return 0;
}

//int main() {
//    data_t data = readfile("data.txt");
//    data_t result;
//    hipMalloc(&result, sizeof(data_t));
//
//    addResult <<<2, 32>>> (&data, &result);
//
//    writeResultToFile(result, "result.txt");
//
//    hipFree(result);
//    for (int i = 0; i < data.numStudents; i++) {
//        free(data.students[i].name);
//    }
//
//    return 0;
//}
/* data_t* result;
    hipMalloc(&result, sizeof(data_t));
    hipMemset(result, 0, sizeof(data_t));*/

    //addResult <<<2, 32>>> (data,result);

   /* data_t h_result;
    hipMemcpy(&h_result, result, sizeof(data_t), hipMemcpyDeviceToHost);*/
    /* for (int i = 0; i < h_result.numStudents; i++) {
           Student student = h_result.students[i];
           std::cout << student.name << ", " << student.year << ", " << student.grade << std::endl;
           free(student.name);
       }*/